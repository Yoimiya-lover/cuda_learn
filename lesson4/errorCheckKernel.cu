#include "hip/hip_runtime.h"
#pragma once

#include <stdio.h>
#include "../tool/common.cuh"

/*************************** 

line93：grid最多指定1024，这里声明了2048个，核函数调用报错

****************************/


__device__ float add(float fpDevice_A,float fpDevice_B)
{
    return fpDevice_A + fpDevice_B;
}

__global__ void MatrixSum1D_GPU(float *fpDevice_A,float *fpDevice_B,float *fpDevice_C,int iElemCount)
{
    int iThreadID = threadIdx.x + blockDim.x * blockIdx.x;
    
    if(iThreadID >= iElemCount) return;
    fpDevice_C[iThreadID] = add(fpDevice_A[iThreadID],fpDevice_B[iThreadID]);
    
}

void MatrixSum1D_CPU(float *fpHost_A,float *fpHost_B,float *fpHost_C,int iElemCount)
{
    for(int i = 0;i < iElemCount;i++)
    {
        fpHost_C[i] = fpHost_A[i] + fpHost_B[i];
    }
}

void initialData(float *addr,int elemCount)
{
    for(int i = 0;i < elemCount;i++)
    {
        addr[i] = (float)(rand()%100)/10.0f;
    }
}


int main(void)
{
    //1.设置GPU设备
    setGPU();

    //2.分配主机内存和设备内存并初始化
    int iElemCount = 1024;                               //设置元素个数
    size_t stBytesCount = iElemCount * sizeof(float);   //设置字节数

    //分配主机内存初始化
    float *fpHost_A,*fpHost_B,*fpHost_C;
    fpHost_A = (float *)malloc(stBytesCount);
    fpHost_B = (float *)malloc(stBytesCount);
    fpHost_C = (float *)malloc(stBytesCount);
    if(fpHost_A != NULL && fpHost_B != NULL && fpHost_C != NULL)
    {
        memset(fpHost_A,0,stBytesCount);
        memset(fpHost_B,0,stBytesCount);
        memset(fpHost_C,0,stBytesCount);
    }
    else
    {
        printf("malloc failed!\n");
        exit(-1);
    }

    // 分配设备内存并初始化
    float *fpDevice_A,*fpDevice_B,*fpDevice_C;
    ErrorCheck(hipMalloc((float**)&fpDevice_A,stBytesCount),__FILE__,__LINE__);
    ErrorCheck(hipMalloc((float**)&fpDevice_B,stBytesCount),__FILE__,__LINE__);
    ErrorCheck(hipMalloc((float**)&fpDevice_C,stBytesCount),__FILE__,__LINE__);
    
    if(fpDevice_A != NULL && fpDevice_B != NULL && fpDevice_C != NULL)
    {
        ErrorCheck(hipMemset(fpDevice_A,0,stBytesCount),__FILE__,__LINE__);
        ErrorCheck(hipMemset(fpDevice_B,0,stBytesCount),__FILE__,__LINE__);
        ErrorCheck(hipMemset(fpDevice_C,0,stBytesCount),__FILE__,__LINE__);
       
    }
    else
    {
        printf("hipMalloc failed!\n");
        exit(-1);
    }
    //初始化主机中数据
    srand(666);//设置随机种子
    initialData(fpHost_A,iElemCount);
    initialData(fpHost_B,iElemCount);

    //数据从主机复制到设备
    ErrorCheck(hipMemcpy(fpDevice_A,fpHost_A,stBytesCount,hipMemcpyHostToDevice),__FILE__,__LINE__);
    ErrorCheck(hipMemcpy(fpDevice_B,fpHost_B,stBytesCount,hipMemcpyHostToDevice),__FILE__,__LINE__);

    //调用核函数在设备进行计算
    dim3 block(2048);//设置线程块大小,最大为1024
    dim3 grid((iElemCount + block.x - 1 )/ block.x);//设置线程块数量,向上取整

    //调用该行会报错
    MatrixSum1D_GPU<<<grid,block>>>(fpDevice_A,fpDevice_B,fpDevice_C,iElemCount);
    ErrorCheck(hipGetLastError(),__FILE__,__LINE__);
    ErrorCheck(hipDeviceSynchronize(),__FILE__,__LINE__);

    //计算数据从设备传回主机
    ErrorCheck(hipMemcpy(fpHost_C,fpDevice_C,stBytesCount,hipMemcpyDeviceToHost),__FILE__,__LINE__);

    for(int i = 0;i < 10;i++)
    {
        printf("idx=%2d\tmatrix_A:%.2f\tmatrix_B:%.2f\tmatrix_C:%.2f\n",i+1,fpHost_A[i],fpHost_B[i],fpHost_C[i]);

    }

    //释放主机与设备内存
    free(fpHost_A);
    free(fpHost_B);
    free(fpHost_C);
    ErrorCheck(hipFree(fpDevice_A),__FILE__,__LINE__);
    ErrorCheck(hipFree(fpDevice_B),__FILE__,__LINE__);
    ErrorCheck(hipFree(fpDevice_C),__FILE__,__LINE__);
    
    //重置
    ErrorCheck(hipDeviceReset(),__FILE__,__LINE__);
    return 0;
}