#pragma once
#include <stdio.h>
#include "../tool/common.cuh"

/**********************************
 * hipMalloc,hipMemcpy,hipDeviceReset,hipFree都是返回hipError_t
 * __FILE__是预处理宏，表示当前文件名
 * __LINE__是预处理宏，表示当前行号
 * 该文件为了测试cuda_error的错误检查函数
 * 
************************************/
int main(void)
{
    int iElemCount = 100;
    float* fpHost_A = (float *)malloc(sizeof(float)*iElemCount);
    memset(fpHost_A,0,sizeof(float)*iElemCount);

    float* fpDevice_A ;
    hipError_t error = ErrorCheck(hipMalloc((float**)&fpDevice_A,sizeof(float)*iElemCount),__FILE__,__LINE__);
    hipMemset(fpDevice_A,0,sizeof(float)*iElemCount);
    ErrorCheck(hipMemcpy(fpDevice_A,fpHost_A,sizeof(float)*iElemCount,hipMemcpyDeviceToHost),__FILE__,__LINE__);//这里有问题，hipMemcpyDeviceToHost设备数据传输给主机

    free(fpHost_A);
    ErrorCheck(hipFree(fpDevice_A),__FILE__,__LINE__);
    ErrorCheck(hipDeviceReset(),__FILE__,__LINE__);
    return 0;
}
