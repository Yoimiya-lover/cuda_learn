#pragma once
#include <hip/hip_runtime.h>


#include <stdio.h>


void setGPU(void)
{

    //检测计算机GPU数量
    int iDeviceCount = 0;
    //返回错误代码
    hipError_t error = hipGetDeviceCount(&iDeviceCount);
    if(error != hipSuccess || iDeviceCount == 0)
    {
        printf("No GPU device found!\n");
        exit(-1);
    }
    else
    {
        printf("GPU device count: %d\n", iDeviceCount);
    }

    //设置执行，一块显卡默认id为0
    int iDevice = 0;
    error = hipSetDevice(iDevice);
    if(error != hipSuccess)
    {
        printf("Failed to set GPU device %d!\n", iDevice);
        exit(-1);
    }
    
}

