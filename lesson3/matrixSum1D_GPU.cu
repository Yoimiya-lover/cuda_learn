#include "hip/hip_runtime.h"
#pragma once

#include <stdio.h>
#include "DeviceCount.cu"

__device__ float add(float fpDevice_A,float fpDevice_B)
{
    return fpDevice_A + fpDevice_B;
}

__global__ void MatrixSum1D_GPU(float *fpDevice_A,float *fpDevice_B,float *fpDevice_C,int iElemCount)
{
    int iThreadID = threadIdx.x + blockDim.x * blockIdx.x;
    
    if(iThreadID >= iElemCount) return;
    fpDevice_C[iThreadID] = add(fpDevice_A[iThreadID],fpDevice_B[iThreadID]);
    
}

void MatrixSum1D_CPU(float *fpHost_A,float *fpHost_B,float *fpHost_C,int iElemCount)
{
    for(int i = 0;i < iElemCount;i++)
    {
        fpHost_C[i] = fpHost_A[i] + fpHost_B[i];
    }
}

void initialData(float *addr,int elemCount)
{
    for(int i = 0;i < elemCount;i++)
    {
        addr[i] = (float)(rand()%100)/10.0f;
    }
}


int main(void)
{
    //1.设置GPU设备
    setGPU();

    //2.分配主机内存和设备内存并初始化
    int iElemCount = 512;                               //设置元素个数
    size_t stBytesCount = iElemCount * sizeof(float);   //设置字节数

    //分配主机内存初始化
    float *fpHost_A,*fpHost_B,*fpHost_C;
    fpHost_A = (float *)malloc(stBytesCount);
    fpHost_B = (float *)malloc(stBytesCount);
    fpHost_C = (float *)malloc(stBytesCount);
    if(fpHost_A != NULL && fpHost_B != NULL && fpHost_C != NULL)
    {
        memset(fpHost_A,0,stBytesCount);
        memset(fpHost_B,0,stBytesCount);
        memset(fpHost_C,0,stBytesCount);
    }
    else
    {
        printf("malloc failed!\n");
        exit(-1);
    }

    // 分配设备内存并初始化
    float *fpDevice_A,*fpDevice_B,*fpDevice_C;
    hipMalloc((float**)&fpDevice_A,stBytesCount);
    hipMalloc((float**)&fpDevice_B,stBytesCount);
    hipMalloc((float**)&fpDevice_C,stBytesCount);
    if(fpDevice_A != NULL && fpDevice_B != NULL && fpDevice_C != NULL)
    {
       hipMemset(fpDevice_A,0,stBytesCount);
       hipMemset(fpDevice_B,0,stBytesCount);
       hipMemset(fpDevice_C,0,stBytesCount);
    }
    else
    {
        printf("hipMalloc failed!\n");
        exit(-1);
    }
    //初始化主机中数据
    srand(666);//设置随机种子
    initialData(fpHost_A,iElemCount);
    initialData(fpHost_B,iElemCount);

    //数据从主机复制到设备
    hipMemcpy(fpDevice_A,fpHost_A,stBytesCount,hipMemcpyHostToDevice);
    hipMemcpy(fpDevice_B,fpHost_B,stBytesCount,hipMemcpyHostToDevice);

    //调用核函数在设备进行计算
    dim3 block(32);//设置线程块大小
    dim3 grid((iElemCount + block.x - 1 )/ block.x);//设置线程块数量,向上取整

    MatrixSum1D_GPU<<<grid,block>>>(fpDevice_A,fpDevice_B,fpDevice_C,iElemCount);
    hipDeviceSynchronize();

    //计算数据从设备传回主机
    hipMemcpy(fpHost_C,fpDevice_C,stBytesCount,hipMemcpyDeviceToHost);

    for(int i = 0;i < 10;i++)
    {
        printf("idx=%2d\tmatrix_A:%.2f\tmatrix_B:%.2f\tmatrix_C:%.2f\n",i+1,fpHost_A[i],fpHost_B[i],fpHost_C[i]);

    }

    //释放主机与设备内存
    free(fpHost_A);
    free(fpHost_B);
    free(fpHost_C);
    hipFree(fpDevice_A);
    hipFree(fpDevice_B);
    hipFree(fpDevice_C);

    //重置
    hipDeviceReset();
    return 0;
}