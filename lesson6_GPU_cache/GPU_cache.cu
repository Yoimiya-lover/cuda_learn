#include "hip/hip_runtime.h"
/*
GPU缓存使用

*/

#include <stdio.h>
#include <iostream>
#include "common.cuh"

__global__ void  test_GPU_cache(void)
{

}


int main(int argc,char **argv)
{
    setGPU();

    hipDeviceProp_t prop;
    if(prop.globalL1CacheSupported)
    {
        std::cout<<"GPU支持全局L1缓存"<<std::endl;
    }
    else
    {
        std::cout<<"GPU不支持全局L1缓存"<<std::endl;
    }

    std::cout<<"L2缓存大小"<<prop.l2CacheSize /(1024 * 1024)<<"M"<<std::endl;

    dim3 block(1);
    dim3 grid(1);
    test_GPU_cache<<<grid,block>>>();
    CUDA_CHECK(hipDeviceSynchronize());//设备同步
    CUDA_CHECK(hipDeviceReset());//设备重置

    return 0;

}