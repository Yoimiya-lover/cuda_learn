#include <stdio.h>
#include <hip/hip_runtime.h> 

//cudaGetDeviceProperties真能在主机端使用，不能写在核函数中
// __global__ void check_device()
// {
//     cudaDeviceProp prop;
//     cudaGetDeviceProperties(&prop, 0); // 0 表示第一张 GPU
//     printf("Device Name: %s\n", prop.name);
//     printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
    
// }
int main()
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0); // 0 表示第一张 GPU
    printf("Device Name: %s\n", prop.name);
    printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
}