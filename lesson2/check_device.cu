#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h> 
#include <iostream>

//hipGetDeviceProperties真能在主机端使用，不能写在核函数中
// __global__ void check_device()
// {
//     hipDeviceProp_t prop;
//     hipGetDeviceProperties(&prop, 0); // 0 表示第一张 GPU
//     printf("Device Name: %s\n", prop.name);
//     printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
    
// }
int main()
{
    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, 0); // 0 表示第一张 GPU
    printf("Device Name: %s\n",deviceProps.name);
    printf("Compute Capability: %d.%d\n", deviceProps.major, deviceProps.minor);
    std::cout << "运行GPU设备:" << deviceProps.name << std::endl;
    std::cout << "SM数量：" << deviceProps.multiProcessorCount << std::endl;
    std::cout << "L2缓存大小：" << deviceProps.l2CacheSize / (1024 * 1024) << "M" << std::endl;
    std::cout << "SM最大驻留线程数量：" << deviceProps.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "设备是否支持流优先级：" << deviceProps.streamPrioritiesSupported << std::endl;
    std::cout << "设备是否支持在L1缓存中缓存全局内存：" << deviceProps.globalL1CacheSupported << std::endl;
    std::cout << "设备是否支持在L1缓存中缓存本地内存：" << deviceProps.localL1CacheSupported << std::endl;
    std::cout << "一个SM可用的最大共享内存量：" << deviceProps.sharedMemPerMultiprocessor / 1024  << "KB" << std::endl;
    std::cout << "一个SM可用的32位最大寄存器数量：" << deviceProps.regsPerMultiprocessor / 1024 << "K" << std::endl;
    std::cout << "一个SM最大驻留线程块数量：" << deviceProps.maxBlocksPerMultiProcessor << std::endl;
    std::cout << "GPU内存带宽：" << deviceProps.memoryBusWidth << std::endl;
    std::cout << "GPU内存频率：" << (float)deviceProps.memoryClockRate / (1024 * 1024) << "GHz" << std::endl;

}