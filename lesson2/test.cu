#include <stdio.h>
#include <hip/hip_runtime.h>  // 确保包含 CUDA 运行时库

__global__ void hello_from_gpu()
{
    //内建变量只在核函数有效
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;

    const int id = threadIdx.x + blockDim.x * blockIdx.x;
    //hello_from_gpu<<<2, 4>>>,2为grid_size,4为block_size
    //grid.Dim = 2    
    //block.Dim = 4
    printf("Hello World from the GPU block%d and thread %d global id:%d\n",bid,tid,id);
}

int main(void)
{
    printf("Hello World from the CPU\n");
    hello_from_gpu<<<2, 4>>>();  // 启动 4x4 个线程
    hipError_t err = hipGetLastError(); // 获取 CUDA 错误
    if (err != hipSuccess)
    {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize(); // 确保 GPU 任务执行完成

    return 0;
}
