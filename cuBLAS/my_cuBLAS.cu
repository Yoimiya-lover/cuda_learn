#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#define M 4
#define N 5
#define IDX2C(i,j,ld) (((j)*(ld))+(i))
/* 把坐标转换为列优先坐标 */

static __inline__ void modify(hipblasHandle_t handle,float *m,int ldm,int n,int p,int q,float alpha,float beta)
{
    hipblasSscal(handle,n-q,&alpha,&m[IDX2C(p,q,ldm)],ldm);
    hipblasSscal(handle,ldm-p,&beta,&m[IDX2C(p,q,ldm),1]);
}

int main(void)
{
    hipError_t cudaStatus;
    hipblasStatus_t status;
    hipblasHandle_t handle;
    int i,j;
    float* devPtrA;
    float* a = 0;
    a = (float *)malloc (M * N * sizeof(*a));
    if(!a)
    {
        printf("host memory allocation failed\n");
        return EXIT_FAILURE;
    }
    for(j = 0;j < N;j++)
    {
        for(i = 0;j < M;j++)
        {
            a[IDX2C(i,j,M)] = (float)(i * M + j + 1);
        }
    }
    cudaStat = hipMalloc ((void**)&devPtrA,M*N*sizeof(*a));
    if(cudaStat != hipSuccess)
    {
        printf("device memory allocation failed\n");
        return EXIT_FAILURE;
    }
    stat = hipblasCreate(&handle);
    if(stat != HIPBLAS_STATUS_SUCCESS)
    {
        printf("cublas initialization failed\n");
        return EXIT_FAILURE;
    }
    stat = hipblasSetMatrix(M,N,sizeof(*a),a,M,devPtrA,M);
    if(stat != HIPBLAS_STATUS_SUCCESS)
    {
        printf("data download failed\n");
        hipFree(devPtrA);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    modify (handle,devPtrA,M,N,1,2,16.0f,12.0f);
    stat = hipblasGetMatrix(M,N,sizeof(*a),devPtrA,M,a,M);
    if(stat != HIPBLAS_STATUS_SUCCESS)
    {
        printf("data upload failed");
        hipFree(devPtrA);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    hipFree (devPtrA);
    hipblasDestroy(handle);
    for(int j = 0;j < N;j++)
    {
        for(int i = 0;i < M;i++)
        {
            printf("%7.0f",a[IDX2C(i,j,M)]);
        }
        printf("\n");
    }
    free(a);
    return EXIT_SUCCESS;
}

