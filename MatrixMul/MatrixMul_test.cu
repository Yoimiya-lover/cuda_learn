/*
*/

#include <iostream>
#include <stdio.h>
#include "MatrixClass.cuh"
#include "MatrixAlgorith.cuh"
#include "tool.cuh"

#define ROLL_NUM 200

int main(int argc,const char* argv[])
{
    if(argc < 4) {
        std::cout << "参数设置错误, 至少需要3个参数!!!" << std::endl;
        return -1;
    }
    const int M = atoi(argv[1]);
    const int K = atoi(argv[2]);
    const int N = atoi(argv[3]);
    if(M <=0 || K <= 0 || N <= 0 ) {
        std::cout << "参数设置错误！！！" << "正确参数为: M > 0; K > 0; N > 0; Thread_num >= 0 !!!" << std::endl;
    }
    setGPU();
    MatrixMul::Matrix<float> mat(M,K,N);
    mat.cudaMem_Host_To_Device();
    dim3 blockSize(32, 32);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x,(M + blockSize.y - 1) / blockSize.y);
    float t_sum = 0;
    for(int i = 0; i < ROLL_NUM; i++)
    {
        hipEvent_t start,stop;
        ErrorCheck(hipEventCreate(&start),__FILE__,__LINE__);
        ErrorCheck(hipEventCreate(&stop),__FILE__,__LINE__);
        ErrorCheck(hipEventRecord(start),__FILE__,__LINE__);
        hipEventQuery(start);

        mat.multiply(gridSize, blockSize);

        ErrorCheck(hipEventRecord(stop),__FILE__,__LINE__);
        ErrorCheck(hipEventSynchronize(stop),__FILE__,__LINE__);
        float elapse_time;
        ErrorCheck(hipEventElapsedTime(&elapse_time,start,stop),__FILE__,__LINE__);

        if(i > 0)
        {
            t_sum += elapse_time;
        }
        ErrorCheck(hipEventDestroy(start),__FILE__,__LINE__);
        ErrorCheck(hipEventDestroy(stop),__FILE__,__LINE__);
        
    }
    const float t_ave = t_sum / ROLL_NUM;
    printf("Average execution time of %d GPU kernel launches = %f (ms)\n",ROLL_NUM,t_ave);

    mat.cudaMem_Device_To_Host();
    mat.check_result();
    mat.MatrixcudaDeviceReset();

    return 0;


}