
#include <hip/hip_runtime.h>
#include <stdio.h>
//#include <cuda_runtime.h>  // 确保包含 CUDA 运行时库

__global__ void hello_from_gpu()
{
    printf("Hello World from the GPU\n");
}

int main(void)
{
    hello_from_gpu<<<4, 4>>>();  // 启动 4x4 个线程
    hipError_t err = hipGetLastError(); // 获取 CUDA 错误
    if (err != hipSuccess)
    {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize(); // 确保 GPU 任务执行完成

    return 0;
}
