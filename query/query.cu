#pragma once
#include <stdio.h>
#include <stdlib.h>
#include "../tool/common.cuh"

int main(void)
{
    int device_id = 0;
    ErrorCheck(hipSetDevice(device_id),__FILE__,__LINE__);
    hipDeviceProp_t prop;
    ErrorCheck(hipGetDeviceProperties(&prop,device_id),__FILE__,__LINE__);
    printf("Device id:%d\n",device_id);
    printf("Device name:%s\n",prop.name);
    printf("Compute capability:d.%d\n",prop.major, prop.minor);
    printf("Amount of global memory:%gGB\n",prop.totalGlobalMem/(1024.0 * 1024.0 * 1024.0));
    printf("Amount of constant memory:%gKB\n",prop.totalConstMem/1024.0);
    printf("Maximum grid size:%d,%d,%d\n",prop.maxGridSize[0],prop.maxGridSize[1],prop.maxGridSize[2]);
    printf("Maximum block size:%d,%d,%d\n",prop.maxThreadsDim[0],prop.maxThreadsDim[1],prop.maxThreadsDim[2]);

    return 0;
}