#include "hip/hip_runtime.h"
/*
 *
 *    静态共享内存使用
 *
 *
*/



#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "common.cuh"

__global__ void test_static_shared_memory(float* d_A,int N)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int n = bid * blockDim.x + tid;
    __shared__ float s_array[32];//共享内存

    if(n < N)
    {
        s_array[tid] = d_A[n];
    }
    __syncthreads();//线程同步

    if(tid == 0)//同一网格共享相同的共享内存，不同网格共享内存不一致
    {
        for(int i = 0;i < 32;i++)
        {
            printf("kernel_1 : %f,blockIdx: %d\n",s_array[i],bid);
        }
    }
}

int main(int argc,char **argv)
{
    setGPU();

    int nElems = 64;
    int nBytes = nElems * sizeof(float);

    float *h_A = (float *)malloc(nBytes);
    for(int i = 0;i < nElems;i++)
    {
        h_A[i] = i;
    }

    float* d_A = nullptr;
    CUDA_CHECK(hipMalloc(&d_A,nBytes));
    CUDA_CHECK(hipMemcpy(d_A,h_A,nBytes,hipMemcpyHostToDevice));

    dim3 block(32);
    dim3 grid(2);
    test_static_shared_memory<<<grid,block>>>(d_A,nElems);

    CUDA_CHECK(hipFree(d_A));
    free(h_A);
    CUDA_CHECK(hipDeviceReset());

}