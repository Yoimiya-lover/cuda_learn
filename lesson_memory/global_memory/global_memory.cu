#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "../tool/common.cuh"

/*静态全局变量使用*/

//静态全局变量必须定义在函数外
__device__ int d_x = 1;
__device__ int d_y[2];

__global__ void test_global_memory(void)
{
    //定义在核函数外，但是核函数可以使用全局变量
    d_y[0] += d_x;
    d_y[1] += d_x;
    printf("d_y[0] = %d,d_y[1] = %d\n",d_y[0],d_y[1]);
}

int main(void)
{
    int devID = 0;
    hipDeviceProp_t deviceProps;
    CUDA_CHECK(hipGetDeviceProperties(&deviceProps,devID));
    std::cout<<"运行GPU设备:"<<deviceProps.name<<std::endl;

    int h_y[2] = {10,20};
    //主机端传输到常量内存
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_y),h_y,sizeof(int) * 2));

    dim3 block(1);
    dim3 grid(1);
    test_global_memory<<<grid,block>>>();
    CUDA_CHECK(hipDeviceSynchronize());
    //常量内存传回主机端
    CUDA_CHECK(hipMemcpyFromSymbol(h_y,HIP_SYMBOL(d_y),sizeof(int) * 2));
    printf("h_y[0] = %d,h_y[1] = %d\n",h_y[0],h_y[1]);

    CUDA_CHECK(hipDeviceReset());

    return 0;


}