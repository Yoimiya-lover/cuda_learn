#include "hip/hip_runtime.h"
/*
 *
 *    动态共享内存使用
 *    核函数外，使用extern修饰符
 *    定义数组时extern __shared__ float s_array[]
 *    调用核函数时，test_dynamic_shared_memory<<<grid,block,32>>>(d_A,nElems);要指定动态内存大小
*/



#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "common.cuh"

extern __shared__ float s_array[];//动态共享内存

__global__ void test_dynamic_shared_memory(float* d_A,int N)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int n = bid * blockDim.x + tid;
   
    if(n < N)
    {
        s_array[tid] = d_A[n];
    }
    __syncthreads();//线程同步

    if(tid == 0)//同一网格共享相同的共享内存，不同网格共享内存不一致
    {
        for(int i = 0;i < 32;i++)
        {
            printf("kernel_1 : %f,blockIdx: %d\n",s_array[i],bid);
        }
    }
}

int main(int argc,char **argv)
{
    setGPU();

    int nElems = 64;
    int nBytes = nElems * sizeof(float);

    float *h_A = (float *)malloc(nBytes);
    for(int i = 0;i < nElems;i++)
    {
        h_A[i] = i;
    }

    float* d_A = nullptr;
    CUDA_CHECK(hipMalloc(&d_A,nBytes));
    CUDA_CHECK(hipMemcpy(d_A,h_A,nBytes,hipMemcpyHostToDevice));

    dim3 block(32);
    dim3 grid(2);
    test_dynamic_shared_memory<<<grid,block,32 * sizeof(float)>>>(d_A,nElems);

    CUDA_CHECK(hipFree(d_A));
    free(h_A);
    CUDA_CHECK(hipDeviceReset());

}