#include "hip/hip_runtime.h"
/*
 *
 *    常量内存使用
 *
 *
*/



#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "common.cuh"

__constant__ float c_data;
__constant__ float c_data2 = 6.6f;

__global__ void test_constant_memory(void)
{
    printf("Constant c_data = %.2f\n",c_data);
}

int main(int argc,char **argv)
{
    setGPU();

    float h_data = 8.8f;
   
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_data),&h_data,sizeof(float)));

    dim3 block(1);
    dim3 grid(1);
    test_constant_memory<<<grid,block>>>();
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipMemcpyFromSymbol(&h_data,HIP_SYMBOL(c_data2),sizeof(float)));
    printf("Constant c_data2 = %.2f\n",h_data);
    
    CUDA_CHECK(hipDeviceReset());

}